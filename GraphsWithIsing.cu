#include<iostream>
#include<fstream>
#include<hip/hip_runtime.h>

using namespace std;

#define cMM hipMallocManaged

class Edge
{
   public:
   int v0,v1,wt;
   __device__ __host__ Edge()
   {
      v0=0,v1=0,wt=0;
   }
};

class Vertices
{
   public:
   int *Neigh;
   Vertices()
   {
      Neigh=nullptr;
   }
};

int main()
{
   int V,E;
   ifstream Graph;
   Graph.open("G13.txt");

   Graph >> V >> E;

   Edge *edges;

   int i= cMM(&edges, E*sizeof(Edge));
   for(i=0;i<E;i++)
   {
      Graph >> edges[i].v0 >> edges[i].v1 >> edges[i].wt;
      cout << edges[i].v0 << " " << edges[i].v1 << " " << edges[i].wt<< "\n" ;
   }
   return 0;
}