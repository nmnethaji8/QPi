#include<iostream>
#include<fstream>
#include<hip/hip_runtime.h>

using namespace std;

#define cMM hipMallocManaged

class Edge
{
   public:
   int v0,v1,wt;
   __device__ __host__ Edge()
   {
      v0=0,v1=0,wt=0;
   }
};

class Vertix
{
   public:
   int *Neigh,n;
   Vertix()
   {
      Neigh=nullptr;
      n=0;
   }
};

int main()
{
   int V,E;
   ifstream Graph;
   Graph.open("G13.txt");

   Graph >> V >> E;

   Edge *edges;

   int i= cMM(&edges, E*sizeof(Edge));
   for(i=0;i<E;i++)
   {
      Graph >> edges[i].v0 >> edges[i].v1 >> edges[i].wt;
      //cout << edges[i].v0 << " " << edges[i].v1 << " " << edges[i].wt<< "\n" ;
   }

   Vertix *vertices;
   i= cMM(&vertices, V*sizeof(Vertix));
   for(i=0;i<E;i++)
   {
      vertices[edges[i].v0-1].n++;
      vertices[edges[i].v1-1].n++;
   }
   for(int j=0;j<V;j++)
   {
      //cout << vertices[j].n << "\n";
      i=cMM(&vertices[j].Neigh, (vertices[j].n)*sizeof(Vertix));
   }
   return 0;
}