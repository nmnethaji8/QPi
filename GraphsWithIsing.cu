#include<iostream>
#include<fstream>
#include<hip/hip_runtime.h>
#include <thrust/random.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_int_distribution.h>

using namespace std;

#define cMM hipMallocManaged

class Edge
{
   public:
   int v0,v1,wt;
   __device__ __host__ Edge()
   {
      v0=0,v1=0,wt=0;
   }
};

class Vertix
{
   public:
   int *Neigh,n,*wt;
   __device__ __host__ Vertix()
   {
      Neigh=nullptr;
      wt=nullptr;
      n=0;
   }
};

int main()
{
   int V,E;
   ifstream Graph;
   Graph.open("G13.txt");

   Graph >> V >> E;

   Edge *edges;

   int i= cMM(&edges, E*sizeof(Edge)),j;
   for(i=0;i<E;i++)
   {
      Graph >> edges[i].v0 >> edges[i].v1 >> edges[i].wt;
      //cout << edges[i].v0 << " " << edges[i].v1 << " " << edges[i].wt<< "\n" ;
   }

//Calculating Number of neighbours for each vertix
   Vertix *vertices;
   i= cMM(&vertices, V*sizeof(Vertix));
   for(i=0;i<E;i++)
   {
      vertices[edges[i].v0-1].n++;
      vertices[edges[i].v1-1].n++;
   }
   for(j=0;j<V;j++)
   {
      //cout << vertices[j].n << "\n";
      i=cMM(&vertices[j].Neigh, (vertices[j].n)*sizeof(int));
      i=cMM(&vertices[j].wt, (vertices[j].n)*sizeof(int));
      vertices[j].n=0;
   }

//Storing Neughbours for each vertix
   for(i=0;i<E;i++)
   {
      vertices[edges[i].v0-1].Neigh[vertices[edges[i].v0-1].n]=edges[i].v1;
      vertices[edges[i].v1-1].Neigh[vertices[edges[i].v1-1].n]=edges[i].v0;

      vertices[edges[i].v0-1].wt[vertices[edges[i].v0-1].n]=edges[i].wt;
      vertices[edges[i].v1-1].wt[vertices[edges[i].v1-1].n]=edges[i].wt;

      vertices[edges[i].v0-1].n+=1;
      vertices[edges[i].v1-1].n+=1;
   }

//Printing Neighbours for each Vertix
   /*for(i=0;i<V;i++)
   {
      cout << i+1 << "\t";
      for(j=0;j<vertices[i].n;j++)
      {
         cout << vertices[i].wt[j] << "\t";
      }
      cout << "\n";
   }*/

//Creating lattice
   thrust::random::ranlux24_base rng;
   thrust::uniform_real_distribution<double> dist(0,1);
   int lattice[V];
   double k;

//Making 75% of the vertices negative
   for(i=0;i<V;i++)
   {
      k=dist(rng);
      if(k<0.75)
      {
         lattice[i]=-1;
      }
      else
      {
         lattice[i]=1;
      }
   }
   return 0;
}