#include<iostream>
#include<fstream>
#include<hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/random.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_int_distribution.h>

using namespace std;

#define cMM hipMallocManaged

template<class T> void print1D( T *array,int const N )
{
   for(int i=0;i<N;i++)
   {
      cout << array[i] << "\n"; 
   }
}

class Edge
{
   public:
   int v0,v1,wt;
   __device__ __host__ Edge()
   {
      v0=0,v1=0,wt=0;
   }
};

class Vertix
{
   public:
   int *Neigh,n,*wt;
   __device__ __host__ Vertix()
   {
      Neigh=nullptr;
      wt=nullptr;
      n=0;
   }
};

int get_energy(int *lattice, Vertix *vertices, int const V)
{
   int LocalEnergy=0,GlobalEnergy=0,i,j;
   for(i=0; i<V; i++)
   {
      for(j=0;j<vertices[i].n;j++)
      {
         //cout << vertices[i].wt[j] << "\t";
         LocalEnergy+=vertices[i].wt[j]*lattice[vertices[i].Neigh[j]-1];
         //cout << vertices[i].wt[j] << " " << lattice[vertices[i].Neigh[j]-1] <<"\t";
      }
      //cout << "\n";
      GlobalEnergy+=LocalEnergy*lattice[i];
   }
   return GlobalEnergy;
}

void metropolis(int *net_spins,int *net_energy,int *lattice,Vertix *vertices, int V ,int times, int InEnergy)
{
   int t,x,spin_i,spin_f,E_i,E_f,j,dE, energy=0;

   thrust::random::ranlux24_base rnd;
   thrust::uniform_int_distribution<int> dist(0,V-1);

   for(t=0;t<times;t++)
   {
      x=dist(rnd);

      spin_i =  lattice[x];   //initial spin
      spin_f = -spin_i;       //proposed spin flip

      //compute change in energy
      E_i = 0;
      E_f = 0;

      for(j=0;j<vertices[x].n;j++)
      {
         E_i+=vertices[x].wt[j]*lattice[vertices[x].Neigh[j]-1]*spin_i;
         E_f+=vertices[x].wt[j]*lattice[vertices[x].Neigh[j]-1]*spin_f;
      }

      dE = E_f-E_i;
      if(dE<0)
      {
         lattice[x]=spin_f;
         energy+=dE;
      }

      net_spins[t] = thrust::reduce(thrust::host, lattice, lattice+V, lattice[0]);
      net_energy[t]= energy;
   }

}

int main()
{
   int V,E;
   ifstream Graph;
   Graph.open("G13.txt");

   Graph >> V >> E;

   Edge *edges;

   int i= cMM(&edges, E*sizeof(Edge)),j;
   for(i=0;i<E;i++)
   {
      Graph >> edges[i].v0 >> edges[i].v1 >> edges[i].wt;
      //cout << edges[i].v0 << " " << edges[i].v1 << " " << edges[i].wt<< "\n" ;
   }

//Calculating Number of neighbours for each vertix
   Vertix *vertices;
   i= cMM(&vertices, V*sizeof(Vertix));
   for(i=0;i<E;i++)
   {
      vertices[edges[i].v0-1].n++;
      vertices[edges[i].v1-1].n++;
   }
   for(j=0;j<V;j++)
   {
      //cout << vertices[j].n << "\n";
      i=cMM(&vertices[j].Neigh, (vertices[j].n)*sizeof(int));
      i=cMM(&vertices[j].wt, (vertices[j].n)*sizeof(int));
      vertices[j].n=0;
   }

//Storing Neughbours for each vertix
   for(i=0;i<E;i++)
   {
      vertices[edges[i].v0-1].Neigh[vertices[edges[i].v0-1].n]=edges[i].v1;
      vertices[edges[i].v1-1].Neigh[vertices[edges[i].v1-1].n]=edges[i].v0;

      vertices[edges[i].v0-1].wt[vertices[edges[i].v0-1].n]=edges[i].wt;
      vertices[edges[i].v1-1].wt[vertices[edges[i].v1-1].n]=edges[i].wt;

      vertices[edges[i].v0-1].n+=1;
      vertices[edges[i].v1-1].n+=1;
   }

//Printing Neighbours for each Vertix
   /*for(i=0;i<V;i++)
   {
      cout << i+1 << "\t";
      for(j=0;j<vertices[i].n;j++)
      {
         cout << vertices[i].wt[j] << "\t";
      }
      cout << "\n";
   }*/

//Creating lattice
   thrust::random::ranlux24_base rng;
   thrust::uniform_real_distribution<double> dist(0,1);
   int *lattice;
   i=cMM(&lattice,V*sizeof(int));
   double k;

//Making 50% of the vertices positive
   for(i=0;i<V;i++)
   {
      k=dist(rng);
      if(k<0.50)
      {
         lattice[i]=-1;
      }
      else
      {
         lattice[i]=1;
      }
   }

//Calculating Energy
   //cout << "Energy of System is "<< get_energy(lattice,vertices,V) <<"\n";

//Calling Metropolis Algorithm
   int *net_spins,*net_energy,times=10000;
   i=cMM(&net_spins,times*sizeof(int));
   i=cMM(&net_energy,times*sizeof(int));
   metropolis(net_spins, net_energy,lattice,vertices, V ,times, get_energy(lattice,vertices,V));

//print1D<int>(net_energy,times);

//Calculating the Best Cut
   int BestCut=0;
   for(i=0;i<E;i++)
   {
      if(lattice[edges[i].v0-1]!=lattice[edges[i].v1-1])
      {
         BestCut++;
      }
   }
   cout <<"The best cut is\t" << BestCut << "\n";
   return 0;
}
