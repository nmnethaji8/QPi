#include<iostream>
#include<fstream>
#include<hip/hip_runtime.h>
#include <thrust/random.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_int_distribution.h>

using namespace std;

#define cMM hipMallocManaged

class Edge
{
   public:
   int v0,v1,wt;
   __device__ __host__ Edge()
   {
      v0=0,v1=0,wt=0;
   }
};

class Vertix
{
   public:
   int *Neigh,n,*wt;
   __device__ __host__ Vertix()
   {
      Neigh=nullptr;
      wt=nullptr;
      n=0;
   }
};

int get_energy(int *lattice, Vertix *vertices, int const V)
{
   int LocalEnergy=0,GlobalEnergy=0,i,j;
   for(i=0; i<V; i++)
   {
      for(j=0;j<vertices[i].n;j++)
      {
         //cout << vertices[i].wt[j] << "\t";
         LocalEnergy+=vertices[i].wt[j]*lattice[vertices[i].Neigh[j]-1];
         //cout << vertices[i].wt[j] << " " << lattice[vertices[i].Neigh[j]-1] <<"\t";
      }
      //cout << "\n";
      GlobalEnergy+=LocalEnergy*lattice[i];
   }
   return GlobalEnergy;
}

void metropolis(int *net_spins,int *net_energy,int *lattice,int V ,int times, int InEnergy)
{
   thrust::random::ranlux24_base rand;
   thrust::uniform_int_distribution<int> dist(0,V-1);

}

int main()
{
   int V,E;
   ifstream Graph;
   Graph.open("G13.txt");

   Graph >> V >> E;

   Edge *edges;

   int i= cMM(&edges, E*sizeof(Edge)),j;
   for(i=0;i<E;i++)
   {
      Graph >> edges[i].v0 >> edges[i].v1 >> edges[i].wt;
      //cout << edges[i].v0 << " " << edges[i].v1 << " " << edges[i].wt<< "\n" ;
   }

//Calculating Number of neighbours for each vertix
   Vertix *vertices;
   i= cMM(&vertices, V*sizeof(Vertix));
   for(i=0;i<E;i++)
   {
      vertices[edges[i].v0-1].n++;
      vertices[edges[i].v1-1].n++;
   }
   for(j=0;j<V;j++)
   {
      //cout << vertices[j].n << "\n";
      i=cMM(&vertices[j].Neigh, (vertices[j].n)*sizeof(int));
      i=cMM(&vertices[j].wt, (vertices[j].n)*sizeof(int));
      vertices[j].n=0;
   }

//Storing Neughbours for each vertix
   for(i=0;i<E;i++)
   {
      vertices[edges[i].v0-1].Neigh[vertices[edges[i].v0-1].n]=edges[i].v1;
      vertices[edges[i].v1-1].Neigh[vertices[edges[i].v1-1].n]=edges[i].v0;

      vertices[edges[i].v0-1].wt[vertices[edges[i].v0-1].n]=edges[i].wt;
      vertices[edges[i].v1-1].wt[vertices[edges[i].v1-1].n]=edges[i].wt;

      vertices[edges[i].v0-1].n+=1;
      vertices[edges[i].v1-1].n+=1;
   }

//Printing Neighbours for each Vertix
   /*for(i=0;i<V;i++)
   {
      cout << i+1 << "\t";
      for(j=0;j<vertices[i].n;j++)
      {
         cout << vertices[i].wt[j] << "\t";
      }
      cout << "\n";
   }*/

//Creating lattice
   thrust::random::ranlux24_base rng;
   thrust::uniform_real_distribution<double> dist(0,1);
   int *lattice;
   i=cMM(&lattice,V*sizeof(int));
   double k;

//Making 50% of the vertices positive
   for(i=0;i<V;i++)
   {
      k=dist(rng);
      if(k<0.50)
      {
         lattice[i]=-1;
      }
      else
      {
         lattice[i]=1;
      }
   }

//Calculating Energy
   //cout << "Energy of System is "<< get_energy(lattice,vertices,V) <<"\n";

//Calling Metropolis Algorithm
   int *net_spins,*net_energy,times=10;
   i=cMM(&net_spins,times*sizeof(int));
   i=cMM(&net_energy,times*sizeof(int));
   metropolis(net_spins, net_energy,lattice, V ,times, get_energy(lattice,vertices,V));

   return 0;
}