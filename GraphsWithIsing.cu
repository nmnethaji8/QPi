#include<iostream>
#include<fstream>
#include<hip/hip_runtime.h>

using namespace std;

#define cMM hipMallocManaged

class Edge
{
   public:
   int v0,v1,wt;
   __device__ __host__ Edge()
   {
      v0=0,v1=0,wt=0;
   }
};

class Vertix
{
   public:
   int *Neigh,n,*wt;
   __device__ __host__ Vertix()
   {
      Neigh=nullptr;
      wt=nullptr;
      n=0;
   }
};

int main()
{
   int V,E;
   ifstream Graph;
   Graph.open("G13.txt");

   Graph >> V >> E;

   Edge *edges;

   int i= cMM(&edges, E*sizeof(Edge)),j,k;
   for(i=0;i<E;i++)
   {
      Graph >> edges[i].v0 >> edges[i].v1 >> edges[i].wt;
      //cout << edges[i].v0 << " " << edges[i].v1 << " " << edges[i].wt<< "\n" ;
   }

//Calculating Number of neighbours for each vertix
   Vertix *vertices;
   i= cMM(&vertices, V*sizeof(Vertix));
   for(i=0;i<E;i++)
   {
      vertices[edges[i].v0-1].n++;
      vertices[edges[i].v1-1].n++;
   }
   for(j=0;j<V;j++)
   {
      //cout << vertices[j].n << "\n";
      i=cMM(&vertices[j].Neigh, (vertices[j].n)*sizeof(int));
      i=cMM(&vertices[j].wt, (vertices[j].n)*sizeof(int));
      vertices[j].n=0;
   }

//Storing Neughbours for each vertix
   for(i=0;i<E;i++)
   {
      vertices[edges[i].v0-1].Neigh[vertices[edges[i].v0-1].n]=edges[i].v1;
      vertices[edges[i].v1-1].Neigh[vertices[edges[i].v1-1].n]=edges[i].v0;

      vertices[edges[i].v0-1].wt[vertices[edges[i].v0-1].n]=edges[i].wt;
      vertices[edges[i].v1-1].wt[vertices[edges[i].v1-1].n]=edges[i].wt;

      vertices[edges[i].v0-1].n+=1;
      vertices[edges[i].v1-1].n+=1;
   }

//Printing Neighbours for each Vertix
   for(i=0;i<V;i++)
   {
      cout << i+1 << "\t";
      for(j=0;j<vertices[i].n;j++)
      {
         cout << vertices[i].wt[j] << "\t";
      }
      cout << "\n";
   }
   return 0;
}