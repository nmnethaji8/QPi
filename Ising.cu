
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

double get_energy(int *lattice, int const N)
{
   double Energy=0,GlobalEnergy=0;
   for(int i=0; i<N; i++)
   {
      for(int j=0; j<N; j++)
      {
         if((i!=0)&&(i!=N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i==0)&&(i!=N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[i*N+j-1];
         }
         if((i!=0)&&(i==N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i!=0)&&(i!=N-1)&&(j==0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j];
         }
         if((i!=0)&&(i!=N-1)&&(j!=0)&&(j==N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i==0)&&(j==0))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1];
         }
         if((i==N-1)&&(j==N-1))
         {
            Energy=lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i==0)&&(j==N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j-1];
         }
         if((i==N-1)&&(j==0))
         {
            Energy=lattice[i*N+j+1]+lattice[(i-1)*N+j];
         }
         GlobalEnergy=GlobalEnergy+(-lattice[i*N+j]*Energy);
      }
   }

   return GlobalEnergy;
}
//-----------------------------------------------------------------
void metropolis(int *net_spins, double *net_energy, int *spin_arr1, int const N, int const times, double const BJ, double const energy)
{
   int spin_arr[N*N];

   for(int i=0; i<N*N;i++)
   {
      spin_arr[i]=spin_arr1[i];
   }

   for(int i=0; i<times;i++)
   {
      net_spins[i]=0;
      net_energy[i]=0;
   }

}

int main()
{
   int lattice_n[]={ 1, -1, -1, -1, -1,  1,  1, 1, -1, -1, 1, -1, -1, -1, -1,  1,  1, -1, -1, -1, -1, -1, -1,  1, -1};
   int lattice_p[]={ 1,  1,  1,  1,  1, -1,  1, 1,  1,  1, 1,  1,  1,  1, -1, -1, -1, -1,  1,  1,  1,  1,  1,  1, -1};
   int N=5;

   cout << "Energy of System is "<< get_energy(lattice_p,N) <<"\t" << get_energy(lattice_n,N)<< endl;

   int *net_spins, times=100;
   double *net_energy;
   net_spins = new int[times];
   net_energy = new double[times];
   metropolis(net_spins, net_energy,lattice_n, N ,times, 0.7, get_energy(lattice_n,N));

   return 0;
}