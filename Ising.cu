
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

double System_Energy(int *lattice, int const N)
{
   double Energy=0,GlobalEnergy=0;
   for(int i=0; i<N; i++)
   {
      for(int j=0; j<N; j++)
      {
         if((i!=0)&&(i!=N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i==0)&&(i!=N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[i*N+j-1];
         }
         if((i!=0)&&(i==N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i!=0)&&(i!=N-1)&&(j==0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j];
         }
         if((i!=0)&&(i!=N-1)&&(j!=0)&&(j==N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i==0)&&(j==0))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1];
         }
         if((i==N-1)&&(j==N-1))
         {
            Energy=lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i==0)&&(j==N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j-1];
         }
         if((i==N-1)&&(j==0))
         {
            Energy=lattice[i*N+j+1]+lattice[(i-1)*N+j];
         }
         GlobalEnergy=GlobalEnergy+(-lattice[i*N+j]*Energy);
      }
   }

   return GlobalEnergy;
}

int main()
{
   int lattice_n[]={ 1, -1, -1, -1, -1,  1,  1, 1, -1, -1, 1, -1, -1, -1, -1,  1,  1, -1, -1, -1, -1, -1, -1,  1, -1};
   int lattice_p[]={ 1,  1,  1,  1,  1, -1,  1, 1,  1,  1, 1,  1,  1,  1, -1, -1, -1, -1,  1,  1,  1,  1,  1,  1, -1};
   int N=5;

   cout << "Energy of System is "<< System_Energy(lattice_p,N) << endl;

   return 0;
}