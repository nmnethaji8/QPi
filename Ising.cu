#include "hip/hip_runtime.h"
#include<iostream>
#include <thrust/random.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_int_distribution.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

using namespace std;

double get_energy(int *lattice, int const N)
{
   double Energy=0,GlobalEnergy=0;
   for(int i=0; i<N; i++)
   {
      for(int j=0; j<N; j++)
      {
         if((i!=0)&&(i!=(N-1))&&(j!=0)&&(j!=(N-1)))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i==0)&&(i!=N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[i*N+j-1];
         }
         if((i!=0)&&(i==N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i!=0)&&(i!=N-1)&&(j==0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j];
         }
         if((i!=0)&&(i!=N-1)&&(j!=0)&&(j==N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i==0)&&(j==0))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1];
         }
         if((i==N-1)&&(j==N-1))
         {
            Energy=lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i==0)&&(j==N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j-1];
         }
         if((i==N-1)&&(j==0))
         {
            Energy=lattice[i*N+j+1]+lattice[(i-1)*N+j];
         }
         GlobalEnergy=GlobalEnergy+(-lattice[i*N+j]*Energy);
      }
   }

   return GlobalEnergy;
}
//-----------------------------------------------------------------
void metropolis(int *net_spins, double *net_energy, int *spin_arr1, int const N, int const times, double const BJ, double const energy)
{
   int spin_arr[N*N],x,y,spin_i,spin_f,E_i,E_f,dE;
   // create a uniform_int_distribution to produce ints from [-7,13]
   thrust::random::ranlux24_base rng, rng2;
   thrust::uniform_int_distribution<int> dist(0,N-1);
   thrust::uniform_int_distribution<double> dist2(0,1);
   for(int i=0; i<N*N;i++)
   {
      spin_arr[i]=spin_arr1[i];
   }

   for(int i=0; i<times;i++)
   {
      net_spins[i]=0;
      net_energy[i]=0;
   }

   for(int t=0;t<times;t++)
   {  
      x=dist(rng);
      y=dist(rng);
      //cout << "(" << x << "," << y << ")\t";

      spin_i = spin_arr[x*N+y]; //initial spin
      spin_f = spin_i*-1; //proposed spin flip

      //compute change in energy
      E_i = 0;
      E_f = 0;
      if(x>0)
      {
         E_i += -spin_i*spin_arr[(x-1)*N+y];
         E_f += -spin_f*spin_arr[(x-1)*N+y];
      }
      if(x<N-1)
      {
         E_i += -spin_i*spin_arr[(x+1)*N+y];
         E_f += -spin_f*spin_arr[(x+1)*N+y];
      }
      if(y>0)
      {
         E_i += -spin_i*spin_arr[x*N+y-1];
         E_f += -spin_f*spin_arr[x*N+y-1];
      }
      if(y<N-1)
      {
         E_i += -spin_i*spin_arr[x*N+y+1];
         E_f += -spin_f*spin_arr[x*N+y+1];
      }

      // 3 / 4. change state with designated probabilities
      /*dE = E_f-E_i
      if((dE>0)&&( dst2(rng2) < exp(-BJ*dE))
      {
         spin_arr[x*y]=spin_f
         energy += dE
      }
      else if(dE<=0)
            spin_arr[x,y]=spin_f
            energy += dE
            
        net_spins[t] = spin_arr.sum()
        net_energy[t] = energy*/
   }
}

int main()
{
   int lattice_n[]={ 1, -1, -1, -1, -1,  1,  1, 1, -1, -1, 1, -1, -1, -1, -1,  1,  1, -1, -1, -1, -1, -1, -1,  1, -1};
   int lattice_p[]={ 1,  1,  1,  1,  1, -1,  1, 1,  1,  1, 1,  1,  1,  1, -1, -1, -1, -1,  1,  1,  1,  1,  1,  1, -1};
   int N=5;

   cout << "Energy of System is "<< get_energy(lattice_p,N) <<"\t" << get_energy(lattice_n,N)<< endl;

   int *net_spins, times=100;
   double *net_energy;
   net_spins = new int[times];
   net_energy = new double[times];
   metropolis(net_spins, net_energy,lattice_n, N ,times, 0.7, get_energy(lattice_n,N));

   return 0;
}