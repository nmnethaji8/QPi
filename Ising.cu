
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

double System_Energy(int *lattice, int N)
{
   double Energy=0,a,b,c,d;
   int lattice2[N*N];
   for(int i=0; i<N; i++)
   {
      for(int j=0; j<N; j++)
      {
         if((i!=0)&&(i!=N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
            cout << lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1] << "\t";
         }
         if((i==0)&&(i!=N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[i*N+j-1];
            cout << lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[i*N+j-1] << "\t";
         }
         if((i!=0)&&(i==N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
            cout << lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1] << "\t";
         }
         if((i!=0)&&(i!=N-1)&&(j==0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j];
            cout << lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j]<< "\t";
         }
         if((i!=0)&&(i!=N-1)&&(j!=0)&&(j==N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
            cout << lattice[(i+1)*N+j]+lattice[(i-1)*N+j]+lattice[i*N+j-1] << "\t";
         }
         if((i==0)&&(j==0))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1];
            cout << lattice[(i+1)*N+j]+lattice[i*N+j+1]<< "\t";
         }
         if((i==N-1)&&(j==N-1))
         {
            Energy=lattice[(i-1)*N+j]+lattice[i*N+j-1];
            cout << lattice[(i-1)*N+j]+lattice[i*N+j-1] << "\t";
         }
         if((i==0)&&(j==N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j-1];
            cout << lattice[(i+1)*N+j]+lattice[i*N+j-1] << "\t";
         }
         if((i==N-1)&&(j==0))
         {
            Energy=lattice[i*N+j+1]+lattice[(i-1)*N+j];
            cout << lattice[i*N+j+1]+lattice[(i-1)*N+j] << "\t";
         }
         lattice2[i*N+j]=Energy;
      }
      cout << "\n";
   }
   return Energy;
}

int main()
{
   int lattice_n[]={ 1, -1, -1, -1, -1,  1,  1, 1, -1, -1, 1, -1, -1, -1, -1,  1,  1, -1, -1, -1, -1, -1, -1,  1, -1};
   int lattice_p[]={ 1,  1,  1,  1,  1, -1,  1, 1,  1,  1, 1,  1,  1,  1, -1, -1, -1, -1,  1,  1,  1,  1,  1,  1, -1};
   int N=5;

   cout << "Energy of System is \n"<< System_Energy(lattice_p,N) << endl;

   return 0;
}