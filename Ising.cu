
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

double System_Energy(int *lattice, int N)
{
   double Energy=0;
   for(int i=0; i<N; i++)
   {
      for(int j=0; j<N; j++)
      {
         cout << lattice[i*N+j] << "\t";
         //Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
      }
      cout << "\n";
   }
   return 0;
}

int main()
{
   int lattice_n[]={ 1, -1, -1, -1, -1,  1,  1, 1, -1, -1, 1, -1, -1, -1, -1,  1,  1, -1, -1, -1, -1, -1, -1,  1, -1};
   int lattice_p[]={ 1,  1,  1,  1,  1, -1,  1, 1,  1,  1, 1,  1,  1,  1, -1, -1, -1, -1,  1,  1,  1,  1,  1,  1, -1};
   int N=5;

   System_Energy(lattice_p,N);

   return 0;
}