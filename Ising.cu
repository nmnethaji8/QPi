
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

double get_energy(int *lattice, int const N)
{
   double Energy=0,GlobalEnergy=0;
   for(int i=0; i<N; i++)
   {
      for(int j=0; j<N; j++)
      {
         if((i!=0)&&(i!=N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i==0)&&(i!=N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[i*N+j-1];
         }
         if((i!=0)&&(i==N-1)&&(j!=0)&&(j!=N-1))
         {
            Energy=lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i!=0)&&(i!=N-1)&&(j==0)&&(j!=N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j];
         }
         if((i!=0)&&(i!=N-1)&&(j!=0)&&(j==N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i==0)&&(j==0))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j+1];
         }
         if((i==N-1)&&(j==N-1))
         {
            Energy=lattice[(i-1)*N+j]+lattice[i*N+j-1];
         }
         if((i==0)&&(j==N-1))
         {
            Energy=lattice[(i+1)*N+j]+lattice[i*N+j-1];
         }
         if((i==N-1)&&(j==0))
         {
            Energy=lattice[i*N+j+1]+lattice[(i-1)*N+j];
         }
         GlobalEnergy=GlobalEnergy+(-lattice[i*N+j]*Energy);
      }
   }

   return GlobalEnergy;
}
//-----------------------------------------------------------------
void metropolis(int net_spins, double net_energy, int *pin_arr, int const times, double const BJ, double const energy)
{

}

int main()
{
   int lattice_n[]={ 1, -1, -1, -1, -1,  1,  1, 1, -1, -1, 1, -1, -1, -1, -1,  1,  1, -1, -1, -1, -1, -1, -1,  1, -1};
   int lattice_p[]={ 1,  1,  1,  1,  1, -1,  1, 1,  1,  1, 1,  1,  1,  1, -1, -1, -1, -1,  1,  1,  1,  1,  1,  1, -1};
   int N=5;

   cout << "Energy of System is "<< get_energy(lattice_p,N) <<"\t" << get_energy(lattice_n,N)<< endl;

   int net_spins=0;
   double net_energy=0;
   metropolis(net_spins, net_energy,lattice_n,100, 0.7, get_energy(lattice_n,N));

   return 0;
}