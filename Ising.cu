
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

double System_Energy(int *lattice, int N)
{
   double Energy=0,a,b,c,d;
   for(int i=1; i<N-1; i++)
   {
      for(int j=1; j<N-1; j++)
      {

         Energy=Energy+lattice[(i+1)*N+j]+lattice[i*N+j+1]+lattice[(i-1)*N+j]+lattice[i*N+j-1];
      }
   }
   return Energy;
}

int main()
{
   int lattice_n[]={ 1, -1, -1, -1, -1,  1,  1, 1, -1, -1, 1, -1, -1, -1, -1,  1,  1, -1, -1, -1, -1, -1, -1,  1, -1};
   int lattice_p[]={ 1,  1,  1,  1,  1, -1,  1, 1,  1,  1, 1,  1,  1,  1, -1, -1, -1, -1,  1,  1,  1,  1,  1,  1, -1};
   int N=5;

   cout << "Energy of System is "<< System_Energy(lattice_p,N) << endl;

   return 0;
}